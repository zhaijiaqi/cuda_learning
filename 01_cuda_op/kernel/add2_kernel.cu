
#include <hip/hip_runtime.h>
__global__ void MatAdd(float* c,
                            const float* a,
                            const float* b,
                            int n)
{
    // 计算当前线程处理的矩阵元素坐标
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = i*n + j;  // 按行优先
    if (i < n && j < n)
        c[idx] = a[idx] + b[idx];
}

void launch_add2(float* c,
                 const float* a,
                 const float* b,
                 int n) {
    dim3 block(16, 16);
    dim3 grid(n/block.x, n/block.y);

    MatAdd<<<grid, block>>>(c, a, b, n);
}